#include <stdio.h>
#include <hip/hip_runtime.h>

__global__ void convolution1D(float *input, float *kernel, float *output, int inputSize, int kernelSize) {
    int halfKernelSize = kernelSize / 2;
    int idx = blockIdx.x * blockDim.x + threadIdx.x;

    if (idx >= halfKernelSize && idx < inputSize - halfKernelSize) {
        float result = 0.0f;

        for (int k = -halfKernelSize; k <= halfKernelSize; k++) {
            result += input[idx + k] * kernel[halfKernelSize + k];
        }
        output[idx] = result;
    }
}

void hostConvolution(float *input, float *kernel, float *output, int inputSize, int kernelSize) {
    float *d_input, *d_kernel, *d_output;
    int inputBytes = inputSize * sizeof(float);
    int kernelBytes = kernelSize * sizeof(float);
    int outputBytes = inputSize * sizeof(float);

    hipMalloc(&d_input, inputBytes);
    hipMalloc(&d_kernel, kernelBytes);
    hipMalloc(&d_output, outputBytes);

    hipMemcpy(d_input, input, inputBytes, hipMemcpyHostToDevice);
    hipMemcpy(d_kernel, kernel, kernelBytes, hipMemcpyHostToDevice);

    int blockSize = 256; // You can change this to optimize performance
    int gridSize = (inputSize + blockSize - 1) / blockSize;

    convolution1D<<<gridSize, blockSize>>>(d_input, d_kernel, d_output, inputSize, kernelSize);
    hipDeviceSynchronize();

    hipMemcpy(output, d_output, outputBytes, hipMemcpyDeviceToHost);

    hipFree(d_input);
    hipFree(d_kernel);
    hipFree(d_output);
}

int main() {
    const int inputSize = 10;
    const int kernelSize = 3;

    // Input array
    float input[inputSize] = {1, 2, 3, 4, 5, 6, 7, 8, 9, 10};
    // Kernel (for example, a simple averaging filter)
    float kernel[kernelSize] = {1.0f/3, 1.0f/3, 1.0f/3};
    // Output array
    float output[inputSize] = {0};

    hostConvolution(input, kernel, output, inputSize, kernelSize);

    // Print the output
    printf("Input:  ");
    for (int i = 0; i < inputSize; i++) {
        printf("%f ", input[i]);
    }
    printf("\n");

    printf("Kernel: ");
    for (int i = 0; i < kernelSize; i++) {
        printf("%f ", kernel[i]);
    }
    printf("\n");

    printf("Output: ");
    for (int i = 0; i < inputSize; i++) {
        printf("%f ", output[i]);
    }
    printf("\n");

    return 0;
}
